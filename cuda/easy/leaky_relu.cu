#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return; 
    const float alpha = 0.01f;

    if (input[idx] > 0) output[idx] = input[idx];
    else output[idx] = input[idx] * alpha;
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}